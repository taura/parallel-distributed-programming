#include "hip/hip_runtime.h"
/*** com 2 */
#include <assert.h>
#include <stdio.h>

#include "cuda_util.h"

__global__ void cuda_thread_fun(long long * p, int n) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  p[i] = i * i;
}

int main(int argc, char ** argv) {
  int n               = (argc > 1 ? atoi(argv[1]) : 10);
  int thread_block_sz = (argc > 2 ? atoi(argv[2]) : 3);
  int n_thread_blocks = (n + thread_block_sz - 1) / thread_block_sz;

/*** if VER == 1 */
  long long * c = (long long *)malloc(sizeof(long long) * n);
  long long * c_dev;
  check_api_error(hipMalloc(&c_dev, sizeof(long long) * n));
  check_launch_error((cuda_thread_fun<<<n_thread_blocks,thread_block_sz>>>(c_dev, n)));
/*** elif VER == 2 */
  long long * c;
  check_api_error(hipMallocManaged(&c, sizeof(long long) * n));
  check_launch_error((cuda_thread_fun<<<n_thread_blocks,thread_block_sz>>>(c, n)));
/*** endif */
  check_api_error(hipDeviceSynchronize());
/*** if VER == 1 */
  check_api_error(hipMemcpy(c, c_dev, sizeof(long long) * n, hipMemcpyDeviceToHost));
/*** endif */
  for (int i = 0; i < n; i++) {
    printf("c[%d] = %ld\n", i, c[i]);
  }
/*** if VER == 1 */
  free(c);
  check_api_error(hipFree(c_dev));
/*** elif VER == 2 */
  check_api_error(hipFree(c));
/*** endif */
  return 0;
}
